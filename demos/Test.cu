#include <iostream>
#include <typeinfo>
#include <cuMat/Core>
#include <Eigen/Core>
#include <chrono>
#include <vector>

using namespace cuMat;
using namespace std;

int main(int argc, char* args[])
{
    std::vector<int> sizes = { 100, 1000, 10000, 100000, 1000000, 10000000 };
    for (int size : sizes) {
        //cuMat
        VectorXf v(size);
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipDeviceSynchronize();
        hipEventRecord(start, cuMat::Context::current().stream());
        //VectorXf r = v * 5;
        (v * 5).eval();
        hipEventRecord(stop, cuMat::Context::current().stream());
        hipEventSynchronize(stop);
        float elapsedCuMat;
        hipEventElapsedTime(&elapsedCuMat, start, stop);
        //Eigen
        Eigen::VectorXf ve(size);
        auto startE = std::chrono::steady_clock::now();
        //Eigen::VectorXf re = ve * 5;
        (ve * 5).eval();
        auto stopE = std::chrono::steady_clock::now();
        double elapsedEigen = std::chrono::duration_cast<std::chrono::duration<double>>(stopE - startE).count() * 1000;

        cout << "Size: " << size << endl;
        cout << "Time cuMat: " << elapsedCuMat << " ms" << endl;
        cout << "Time Eigen: " << elapsedEigen << " ms" << endl;
    }
}