#include "benchmark.h"

#include <Eigen/Sparse>
#include <cuMat/Core>
#include <cuMat/Sparse>
#include <iostream>
#include <cstdlib>

void benchmark_cuMat(
    const std::vector<std::string>& parameterNames,
    const Json::Array& parameters,
    const std::vector<std::string>& returnNames,
    Json::Array& returnValues)
{
    //number of runs for time measures
    const int runs = 10;

    int numConfigs = parameters.Size();
    for (int config = 0; config < numConfigs; ++config)
    {
		//Input
		int gridSize = parameters[config][0].AsInt32();
		double totalTime = 0;
		std::cout << "  Grid Size: " << gridSize << std::flush;
		int matrixSize = gridSize * gridSize;

		//Create matrix
#define IDX(x, y) ((y) + (x)*gridSize)
		Eigen::SparseMatrix<float, Eigen::RowMajor, int> matrix(matrixSize, matrixSize);
		matrix.reserve(Eigen::VectorXi::Constant(matrixSize, 5));
		for (int x = 0; x<gridSize; ++x) for (int y = 0; y<gridSize; ++y)
		{
			int row = IDX(x, y);
			if (x > 0) matrix.insert(row, IDX(x - 1, y)) = -1;
			if (y > 0) matrix.insert(row, IDX(x, y - 1)) = -1;
			matrix.insert(row, row) = 4;
			if (y < gridSize - 1) matrix.insert(row, IDX(x, y + 1)) = -1;
			if (x < gridSize - 1) matrix.insert(row, IDX(x + 1, y)) = -1;
		}
		matrix.makeCompressed();

		//Create vector
		Eigen::VectorXf ex = Eigen::VectorXf::Random(matrixSize);

		//Send to cuMat
		typedef cuMat::SparseMatrix<float, 1, cuMat::CSR> SMatrix;
		cuMat::SparsityPattern pattern;
		pattern.rows = matrixSize;
		pattern.cols = matrixSize;
		pattern.nnz = matrix.nonZeros();
		pattern.JA = SMatrix::IndexVector(matrixSize + 1); pattern.JA.copyFromHost(matrix.outerIndexPtr());
		pattern.IA = SMatrix::IndexVector(pattern.nnz); pattern.IA.copyFromHost(matrix.innerIndexPtr());
        pattern.assertValid<cuMat::CSR>();
		SMatrix mat(pattern);
		mat.getData().copyFromHost(matrix.valuePtr());

		cuMat::VectorXf x = cuMat::VectorXf::fromEigen(ex);
		cuMat::VectorXf r(matrixSize);

        //Run it multiple times
        for (int run = 0; run < runs; ++run)
        {
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);

            //Main logic
			hipDeviceSynchronize();
			//hipEventRecord(start, cuMat::Context::current().stream());
			auto start2 = std::chrono::steady_clock::now();

			for (int i = 0; i < 10; ++i) {
				r.inplace() = mat * x;
			}

			//hipEventRecord(stop, cuMat::Context::current().stream());
			//hipEventSynchronize(stop);
			//float elapsed;
			//hipEventElapsedTime(&elapsed, start, stop);

			hipDeviceSynchronize();
			auto finish2 = std::chrono::steady_clock::now();
			double elapsed = std::chrono::duration_cast<
				std::chrono::duration<double> >(finish2 - start2).count() * 1000;

            totalTime += elapsed;
            hipEventDestroy(start);
            hipEventDestroy(stop);
        }

        //Result
        Json::Array result;
        double finalTime = totalTime / runs;
        result.PushBack(finalTime);
        returnValues.PushBack(result);
        std::cout << " -> " << finalTime << "ms" << std::endl;
    }
}