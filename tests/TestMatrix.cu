#include "hip/hip_runtime.h"
#include <catch/catch.hpp>

#include <cuMat/src/Matrix.h>
#include <cuMat/src/EigenInteropHelpers.h>

#define TEST_SIZE_F1(type, flags, rowCompile, rowRuntime, colCompile, colRuntime, batchCompile, batchRuntime) \
	do{ \
	cuMat::Matrix<type, rowCompile, colCompile, batchCompile, flags> m(rowRuntime, colRuntime, batchRuntime); \
	REQUIRE(m.rows() == rowRuntime); \
	REQUIRE(m.cols() == colRuntime); \
	REQUIRE(m.batches() == batchRuntime); \
	REQUIRE(m.size() == rowRuntime*colRuntime*batchRuntime); \
	if (m.size()>0) REQUIRE(m.data() != nullptr); \
	}while(false)

#define TEST_SIZE_F2(rowCompile, rowRuntime, colCompile, colRuntime, batchCompile, batchRuntime) \
	do{ \
	TEST_SIZE_F1(bool, cuMat::RowMajor,   rowCompile, rowRuntime, colCompile, colRuntime, batchCompile, batchRuntime); \
	TEST_SIZE_F1(bool, cuMat::ColumnMajor,rowCompile, rowRuntime, colCompile, colRuntime, batchCompile, batchRuntime); \
	TEST_SIZE_F1(int, cuMat::RowMajor,   rowCompile, rowRuntime, colCompile, colRuntime, batchCompile, batchRuntime); \
	TEST_SIZE_F1(int, cuMat::ColumnMajor,rowCompile, rowRuntime, colCompile, colRuntime, batchCompile, batchRuntime); \
	TEST_SIZE_F1(float, cuMat::RowMajor,   rowCompile, rowRuntime, colCompile, colRuntime, batchCompile, batchRuntime); \
	TEST_SIZE_F1(float, cuMat::ColumnMajor,rowCompile, rowRuntime, colCompile, colRuntime, batchCompile, batchRuntime); \
	TEST_SIZE_F1(double, cuMat::RowMajor,   rowCompile, rowRuntime, colCompile, colRuntime, batchCompile, batchRuntime); \
	TEST_SIZE_F1(double, cuMat::ColumnMajor,rowCompile, rowRuntime, colCompile, colRuntime, batchCompile, batchRuntime); \
	}while(false)
	

TEST_CASE("instantiation_fully", "[matrix]")
{
	TEST_SIZE_F2(0, 0, 0, 0, 0, 0);

	TEST_SIZE_F2(1, 1, 1, 1, 1, 1);
	TEST_SIZE_F2(8, 8, 1, 1, 1, 1);
	TEST_SIZE_F2(1, 1, 8, 8, 1, 1);
	TEST_SIZE_F2(1, 1, 1, 1, 8, 8);
	TEST_SIZE_F2(8, 8, 8, 8, 1, 1);
	TEST_SIZE_F2(8, 8, 1, 1, 8, 8);
	TEST_SIZE_F2(1, 1, 8, 8, 8, 8);

	TEST_SIZE_F2(cuMat::Dynamic, 16, 4, 4, 4, 4);
	TEST_SIZE_F2(4, 4, cuMat::Dynamic, 16, 4, 4);
	TEST_SIZE_F2(4, 4, 4, 4, cuMat::Dynamic, 16);
	TEST_SIZE_F2(cuMat::Dynamic, 16, cuMat::Dynamic, 8, 4, 4);
	TEST_SIZE_F2(4, 4, cuMat::Dynamic, 16, cuMat::Dynamic, 8);
	TEST_SIZE_F2(cuMat::Dynamic, 8, 4, 4, cuMat::Dynamic, 16);
	TEST_SIZE_F2(cuMat::Dynamic, 8, cuMat::Dynamic, 32, cuMat::Dynamic, 16);
}

#define TEST_SIZE_D1(type, flags, Rows, Cols, Batches) \
	do { \
	cuMat::Matrix<type, Rows, Cols, Batches, flags> m; \
	if (Rows > 0) {\
		REQUIRE(m.rows() == Rows); \
	} else {\
		REQUIRE(m.rows() == 0); \
	} if (Cols > 0) { \
		REQUIRE(m.cols() == Cols); \
	} else {\
		REQUIRE(m.cols() == 0); \
	} if (Batches > 0) { \
		REQUIRE(m.batches() == Batches); \
	} else {\
		REQUIRE(m.batches() == 0); \
	} if (Rows>0 && Cols>0 && Batches>0) { \
		REQUIRE(m.data() != nullptr); \
	} else {\
		REQUIRE(m.data() == nullptr); \
	} \
	} while (false)
#define TEST_SIZE_D2(rows, cols, batches) \
	do { \
	TEST_SIZE_D1(bool, cuMat::RowMajor, rows, cols, batches); \
	TEST_SIZE_D1(bool, cuMat::ColumnMajor, rows, cols, batches); \
	TEST_SIZE_D1(int, cuMat::RowMajor, rows, cols, batches); \
	TEST_SIZE_D1(int, cuMat::ColumnMajor, rows, cols, batches); \
	TEST_SIZE_D1(float, cuMat::RowMajor, rows, cols, batches); \
	TEST_SIZE_D1(float, cuMat::ColumnMajor, rows, cols, batches); \
	TEST_SIZE_D1(double, cuMat::RowMajor, rows, cols, batches); \
	TEST_SIZE_D1(double, cuMat::ColumnMajor, rows, cols, batches); \
	} while(false)

TEST_CASE("instantiation_default", "[matrix]")
{
	TEST_SIZE_D2(2, 4, 8);
	TEST_SIZE_D2(cuMat::Dynamic, 4, 8);
	TEST_SIZE_D2(2, cuMat::Dynamic, 8);
	TEST_SIZE_D2(2, 4, cuMat::Dynamic);
	TEST_SIZE_D2(cuMat::Dynamic, cuMat::Dynamic, 8);
	TEST_SIZE_D2(cuMat::Dynamic, 4, cuMat::Dynamic);
	TEST_SIZE_D2(2, cuMat::Dynamic, cuMat::Dynamic);
	TEST_SIZE_D2(cuMat::Dynamic, cuMat::Dynamic, cuMat::Dynamic);
}

TEST_CASE("instantiation_vector", "[matrix]")
{
	cuMat::Matrix<float, 1, cuMat::Dynamic, 1, 0> columnV(8);
	REQUIRE(columnV.rows() == 1);
	REQUIRE(columnV.cols() == 8);
	REQUIRE(columnV.batches() == 1);
	cuMat::Matrix<float, cuMat::Dynamic, 1, 1, 0> rowV(8);
	REQUIRE(rowV.rows() == 8);
	REQUIRE(rowV.cols() == 1);
	REQUIRE(rowV.batches() == 1);
}

#define TEST_SIZE_M(rowCompile, rowRuntime, colCompile, colRuntime) \
	do {\
	cuMat::Matrix<float, rowCompile, colCompile, 1, 0> m(rowRuntime, colRuntime); \
	REQUIRE(m.rows() == rowRuntime); \
	REQUIRE(m.cols() == colRuntime); \
	REQUIRE(m.batches() == 1); \
	REQUIRE(m.size() == rowRuntime*colRuntime); \
	} while(0)
TEST_CASE("instantiation_matrix", "[matrix]")
{
	TEST_SIZE_M(4, 4, 8, 8);
	TEST_SIZE_M(cuMat::Dynamic, 4, 8, 8);
	TEST_SIZE_M(4, 4, cuMat::Dynamic, 8);
	TEST_SIZE_M(cuMat::Dynamic, 4, cuMat::Dynamic, 8);
}

TEST_CASE("instantiation_throws", "[matrix]")
{
	cuMat::Context& ctx = cuMat::Context::current();
	REQUIRE_THROWS((cuMat::Matrix<float, 8, 6, 4, 0>(7, 6, 4)));
	REQUIRE_THROWS((cuMat::Matrix<float, 8, 6, 4, 0>(8, 7, 4)));
	REQUIRE_THROWS((cuMat::Matrix<float, 8, 6, 4, 0>(8, 6, 3)));

	REQUIRE_THROWS((cuMat::Matrix<float, cuMat::Dynamic, 6, 4, 0>(-1, 6, 4)));
	REQUIRE_THROWS((cuMat::Matrix<float, 8, cuMat::Dynamic, 4, 0>(8, -1, 4)));
	REQUIRE_THROWS((cuMat::Matrix<float, 8, 6, cuMat::Dynamic, 0>(8, 6, -1)));

	REQUIRE_THROWS((cuMat::Matrix<float, cuMat::Dynamic, cuMat::Dynamic, 4, 0>(-1, 6, 4)));
	REQUIRE_THROWS((cuMat::Matrix<float, cuMat::Dynamic, cuMat::Dynamic, 4, 0>(8, -1, 4)));
	REQUIRE_THROWS((cuMat::Matrix<float, cuMat::Dynamic, 6, cuMat::Dynamic, 0>(-1, 6, 4)));
	REQUIRE_THROWS((cuMat::Matrix<float, cuMat::Dynamic, 6, cuMat::Dynamic, 0>(8, 6, -1)));
	REQUIRE_THROWS((cuMat::Matrix<float, cuMat::Dynamic, 6, cuMat::Dynamic, 0>(8, -1, 4)));
	REQUIRE_THROWS((cuMat::Matrix<float, cuMat::Dynamic, 6, cuMat::Dynamic, 0>(8, 6, -1)));

	REQUIRE_THROWS((cuMat::Matrix<float, cuMat::Dynamic, cuMat::Dynamic, cuMat::Dynamic, 0>(-1, 6, 4)));
	REQUIRE_THROWS((cuMat::Matrix<float, cuMat::Dynamic, cuMat::Dynamic, cuMat::Dynamic, 0>(8, -1, 4)));
	REQUIRE_THROWS((cuMat::Matrix<float, cuMat::Dynamic, cuMat::Dynamic, cuMat::Dynamic, 0>(8, 6, -1)));
}


TEST_CASE("index_computations_rowMajor", "[matrix]")
{
	cuMat::Matrix<int, 5, 16, 7, cuMat::RowMajor> m;
	for (Index i=0; i<m.rows(); ++i)
	{
		for (Index j=0; j<m.cols(); ++j)
		{
			for (Index k=0; k<m.batches(); ++k)
			{
				Index index = m.index(i, j, k);
				REQUIRE(index >= 0);
				REQUIRE(index < m.size());
				Index i2, j2, k2;
				m.index(index, i2, j2, k2);
				REQUIRE(i2 == i);
				REQUIRE(j2 == j);
				REQUIRE(k2 == k);
			}
		}
	}
}
TEST_CASE("index_computations_columnMajor", "[matrix]")
{
	cuMat::Matrix<int, 5, 16, 7, cuMat::ColumnMajor> m;
	for (Index i = 0; i<m.rows(); ++i)
	{
		for (Index j = 0; j<m.cols(); ++j)
		{
			for (Index k = 0; k<m.batches(); ++k)
			{
				Index index = m.index(i, j, k);
				REQUIRE(index >= 0);
				REQUIRE(index < m.size());
				Index i2, j2, k2;
				m.index(index, i2, j2, k2);
				REQUIRE(i2 == i);
				REQUIRE(j2 == j);
				REQUIRE(k2 == k);
			}
		}
	}
}

template<typename MatrixType>
__global__ void TestMatrixWriteRawKernel(dim3 virtual_size, MatrixType matrix)
{
	CUMAT_KERNEL_1D_LOOP(i, virtual_size)
	{
		matrix.rawCoeff(i) = i;
	}
}
//Tests if a kernel can write the raw data
TEST_CASE("write_raw", "[matrix]")
{
	cuMat::Context& ctx = cuMat::Context::current();

	int sx = 4;
	int sy = 8;
	int sz = 16;
	cuMat::Matrix<int, cuMat::Dynamic, cuMat::Dynamic, cuMat::Dynamic, 0> m(sx, sy, sz);

	cuMat::KernelLaunchConfig cfg = ctx.createLaunchConfig1D((unsigned int) m.size());
	TestMatrixWriteRawKernel <<< cfg.block_count, cfg.thread_per_block, 0, ctx.stream() >>>
		(cfg.virtual_size, m);
	CUMAT_CHECK_ERROR();

	std::vector<int> host(sx * sy * sz);
	m.copyToHost(&host[0]);
	for (int i=0; i<sx*sy*sz; ++i)
	{
		REQUIRE(host[i] == i);
	}
}

template<typename MatrixType>
__global__ void TestMatrixReadRawKernel(dim3 virtual_size, MatrixType matrix, int* failure)
{
	CUMAT_KERNEL_1D_LOOP(i, virtual_size)
	{
		if (matrix.rawCoeff(i) != i) failure[0] = 1;
	}
}
//Test if the kernel can read the raw data
TEST_CASE("read_raw", "[matrix]")
{
	cuMat::Context& ctx = cuMat::Context::current();

	int sx = 4;
	int sy = 8;
	int sz = 16;
	cuMat::Matrix<int, cuMat::Dynamic, cuMat::Dynamic, cuMat::Dynamic, 0> m(sx, sy, sz);

	std::vector<int> host1(sx * sy * sz);
	for (int i = 0; i<sx*sy*sz; ++i)
	{
		host1[i] = i;
	}
	m.copyFromHost(host1.data());

	cuMat::DevicePointer<int> successFlag(1);
	CUMAT_SAFE_CALL(hipMemset(successFlag.pointer(), 0, sizeof(int)));

	cuMat::KernelLaunchConfig cfg = ctx.createLaunchConfig1D((unsigned int) m.size());
	TestMatrixReadRawKernel <<< cfg.block_count, cfg.thread_per_block, 0, ctx.stream() >>>
		(cfg.virtual_size, m, successFlag.pointer());
	CUMAT_CHECK_ERROR();

	int successFlagHost;
	hipMemcpy(&successFlagHost, successFlag.pointer(), sizeof(int), hipMemcpyDeviceToHost);
	REQUIRE(successFlagHost == 0);
}


template<typename MatrixType>
__global__ void TestMatrixWriteCoeffKernel(dim3 virtual_size, MatrixType matrix)
{
	CUMAT_KERNEL_3D_LOOP(i, j, k, virtual_size)
	{
		matrix.coeff(i, j, k) = i + j*100 + k * 100*100;
	}
}
//Tests if a kernel can write the 3d-indexed coefficients
TEST_CASE("write_coeff_columnMajor", "[matrix]")
{
	cuMat::Context& ctx = cuMat::Context::current();

	int sx = 4;
	int sy = 8;
	int sz = 16;
	cuMat::Matrix<int, cuMat::Dynamic, cuMat::Dynamic, cuMat::Dynamic, cuMat::ColumnMajor> m(sx, sy, sz);

	cuMat::KernelLaunchConfig cfg = ctx.createLaunchConfig3D(sx, sy, sz);
	TestMatrixWriteCoeffKernel <<< cfg.block_count, cfg.thread_per_block, 0, ctx.stream() >>>
		(cfg.virtual_size, m);
	CUMAT_CHECK_ERROR();

	std::vector<int> host(sx * sy * sz);
	m.copyToHost(&host[0]);
	int i = 0;
	for (int z=0; z<sz; ++z)
	{
		for (int y=0; y<sy; ++y)
		{
			for (int x=0; x<sx; ++x)
			{
				REQUIRE(host[i] == x + y * 100 + z * 100 * 100);
				i++;
			}
		}
	}
}
//Tests if a kernel can write the 3d-indexed coefficients
TEST_CASE("write_coeff_rowMajor", "[matrix]")
{
	cuMat::Context& ctx = cuMat::Context::current();

	int sx = 4;
	int sy = 8;
	int sz = 16;
	cuMat::Matrix<int, cuMat::Dynamic, cuMat::Dynamic, cuMat::Dynamic, cuMat::RowMajor> m(sx, sy, sz);

	cuMat::KernelLaunchConfig cfg = ctx.createLaunchConfig3D(sx, sy, sz);
	TestMatrixWriteCoeffKernel <<< cfg.block_count, cfg.thread_per_block, 0, ctx.stream() >>>
		(cfg.virtual_size, m);
	CUMAT_CHECK_ERROR();

	std::vector<int> host(sx * sy * sz);
	m.copyToHost(&host[0]);
	int i = 0;
	for (int z = 0; z<sz; ++z)
	{
		for (int x = 0; x<sx; ++x)
		{
			for (int y = 0; y<sy; ++y)
			{
				REQUIRE(host[i] == x + y * 100 + z * 100 * 100);
				i++;
			}
		}
	}
}

// EIGEN INTEROP

template<typename _Matrix>
void testMatrixToEigen(const _Matrix& m)
{
	cuMat::Context& ctx = cuMat::Context::current();
	int sx = m.rows();
	int sy = m.cols();
	cuMat::KernelLaunchConfig cfg = ctx.createLaunchConfig3D(sx, sy, 1);
	TestMatrixWriteCoeffKernel <<< cfg.block_count, cfg.thread_per_block, 0, ctx.stream() >>>
		(cfg.virtual_size, m);
	CUMAT_CHECK_ERROR();

	auto host = m.toEigen();
	for (int y = 0; y<sy; ++y)
	{
		for (int x = 0; x<sx; ++x)
		{
			REQUIRE(host(x, y) == x + y * 100);
		}
	}
}

TEST_CASE("matrix_to_eigen", "[matrix]")
{
	testMatrixToEigen(cuMat::Matrix<float, 4, 8, 1, cuMat::ColumnMajor>(4, 8, 1));
	testMatrixToEigen(cuMat::Matrix<int, 16, 8, 1, cuMat::ColumnMajor>(16, 8, 1));
	testMatrixToEigen(cuMat::Matrix<float, cuMat::Dynamic, cuMat::Dynamic, 1, cuMat::ColumnMajor>(32, 6, 1));

	testMatrixToEigen(cuMat::Matrix<float, 4, 8, 1, cuMat::RowMajor>(4, 8, 1));
	testMatrixToEigen(cuMat::Matrix<int, 16, 8, 1, cuMat::RowMajor>(16, 8, 1));
	testMatrixToEigen(cuMat::Matrix<float, cuMat::Dynamic, cuMat::Dynamic, 1, cuMat::RowMajor>(32, 6, 1));
}

template<typename MatrixType>
__global__ void TestMatrixWriteCoeffKernel(dim3 virtual_size, MatrixType matrix, int* failure)
{
	CUMAT_KERNEL_3D_LOOP(i, j, k, virtual_size)
	{
		if (matrix.coeff(i, j, k) != i + j * 100 + k * 100 * 100)
			failure[0] = 1;
	}
}
template <typename _Matrix>
void testMatrixFromEigen(const _Matrix& m)
{
	int sx = m.rows();
	int sy = m.cols();
	_Matrix host = m;

	for (int y = 0; y<sy; ++y)
	{
		for (int x = 0; x<sx; ++x)
		{
			host(x, y) = x + y * 100;
		}
	}

	cuMat::Context& ctx = cuMat::Context::current();

	typedef typename cuMat::eigen::MatrixEigenToCuMat<_Matrix>::type matrix_t;
	matrix_t mat = matrix_t::fromEigen(host);

	cuMat::DevicePointer<int> successFlag(1);
	CUMAT_SAFE_CALL(hipMemset(successFlag.pointer(), 0, sizeof(int)));

	cuMat::KernelLaunchConfig cfg = ctx.createLaunchConfig3D(sx, sy, 1);
	TestMatrixWriteCoeffKernel <<< cfg.block_count, cfg.thread_per_block, 0, ctx.stream() >>>
		(cfg.virtual_size, mat, successFlag.pointer());
	CUMAT_CHECK_ERROR();

	int successFlagHost;
	hipMemcpy(&successFlagHost, successFlag.pointer(), sizeof(int), hipMemcpyDeviceToHost);
	REQUIRE(successFlagHost == 0);
}
TEST_CASE("matrix_from_eigen", "[matrix]")
{
	testMatrixFromEigen(Eigen::Matrix<float, 8, 6, Eigen::RowMajor>());
	{
		auto m = Eigen::Matrix<float, Eigen::Dynamic, 6, Eigen::RowMajor>();
		m.resize(12, 6);
		testMatrixFromEigen(m);
	}
	{
		auto m = Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>();
		m.resize(12, 24);
		testMatrixFromEigen(m);
	}

	testMatrixFromEigen(Eigen::Matrix<float, 8, 6, Eigen::ColMajor>());
	{
		auto m = Eigen::Matrix<float, 16, Eigen::Dynamic, Eigen::ColMajor>();
		m.resize(16, 8);
		testMatrixFromEigen(m);
	}
	{
		auto m = Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::ColMajor>();
		m.resize(12, 24);
		testMatrixFromEigen(m);
	}
}

// Matrix assignments

TEST_CASE("assign", "[matrix]")
{
	cuMat::Matrix<int, 5, 7, 3, cuMat::RowMajor> mat1;
	REQUIRE(mat1.dataPointer().getCounter() == 1);
	
	cuMat::Matrix<int, cuMat::Dynamic, 7, 3, cuMat::RowMajor> mat2(mat1);
	REQUIRE(mat1.dataPointer().getCounter() == 2);
	
	cuMat::Matrix<int, 5, 7, cuMat::Dynamic, cuMat::RowMajor> mat3;
	mat3 = mat1;
	REQUIRE(mat1.dataPointer().getCounter() == 3);

	cuMat::Matrix<int, cuMat::Dynamic, 7, cuMat::Dynamic, cuMat::RowMajor> mat4(mat3);
	REQUIRE(mat1.dataPointer().getCounter() == 4);
	REQUIRE(mat4.dataPointer().getCounter() == 4);
	
	REQUIRE(mat1.data() == mat2.data());
	REQUIRE(mat1.data() == mat3.data());
	REQUIRE(mat1.data() == mat4.data());
}